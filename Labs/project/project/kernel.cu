#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "kernel.h"
#include "utils.h"

#define MAX_THREADS_DIM 32

__global__ void distanceKernel(Point points[], int n, double* distances, int startX, int startY) {
	int x = blockIdx.x * blockDim.x + threadIdx.x + startX;
	int y = blockIdx.y * blockDim.y + threadIdx.y + startY;

	if (x < n && y < n) {
		distances[x + y * n] = distances[x * n + y] = pow(points[x].x - points[y].x, 2) + pow(points[x].y - points[y].y, 2);
	}
}

int runOnCUDA(Point points[], int n, double* distances, int startX, int startY, int winSize) {
	Point *dev_points;
	double *dev_distances, *tmp_distances;
	hipError_t cudaStatus;
	int i, blocks, threads;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 1;
	}

	cudaStatus = hipMalloc((void**)&dev_distances, n * n * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 1;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! 1");
		return 1;
	}

	cudaStatus = hipMemcpy(dev_distances, distances, n * n * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! 2");
		return 1;
	}

	blocks = (int)ceil((double) winSize / MAX_THREADS_DIM);
	threads = (int)fmin((double)MAX_THREADS_DIM, (double)winSize);

	printf("Blocks: %d, Threads: %d", blocks, threads);

	// Launch a kernel on the GPU with one thread for each element.
	distanceKernel << <dim3(blocks, blocks), dim3(threads, threads) >> >(dev_points, n, dev_distances, startX, startY);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "distanceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return 1;
	}

	tmp_distances = (double*) malloc(n * n * sizeof(double));

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(tmp_distances, dev_distances, n * n * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 1;
	}

	hipFree(dev_points);
	hipFree(dev_distances);

	for (i = startY; i < startY + winSize; i++) {
		memcpy(distances + i * n + startX, tmp_distances + i * n + startX, winSize * sizeof(double));
	}

	for (i = startX; i < startX + winSize; i++) {
		memcpy(distances + i * n + startY, tmp_distances + i * n + startY, winSize * sizeof(double));
	}

	free(tmp_distances);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return cudaStatus;
}
