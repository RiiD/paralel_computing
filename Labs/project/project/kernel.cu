#include "hip/hip_runtime.h"
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"
#include "utils.h"
#include "config.h"

static Point *dev_points;
static double *dev_distances;
static hipError_t cudaStatus;

/**
 * Calculates distances from one point to another. Runs on CUDA device.
 * @param const Point* points
 * @param double* distances
 * @param int n
 * @param int startPoint
 */
__global__ void distanceKernel(const Point points[], double distances[], int n, int startPoint) {
	int y = blockIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < n) {
		distances[y * n + x] =
			(points[x].x - points[y + startPoint].x) * 
			(points[x].x - points[y + startPoint].x) + 
			(points[x].y - points[y + startPoint].y) * 
			(points[x].y - points[y + startPoint].y);
	}
}

/**
 * Initializes CUDA device. Allocates memory.
 * @param const Point* points
 * @param int n
 * @param int pointsCount
 * @returns int Cuda status code
 */
int cudaInit(const Point points[], int n, int pointsCount) {

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 1;
	}

	cudaStatus = hipMalloc((void**)&dev_distances, pointsCount * n * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 1;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! 1");
		return 1;
	}

	return cudaStatus;
}

/**
 * Starts calculating distances for given number of points starting from statPoints on CUDA device.
 * @param double* distances
 * @param int n
 * @param int k
 * @param int startPoint
 * @param int pointsCount
 * @returns int Cuda status code
 */
int runOnCUDA(int n, int startPoint, int pointsCount) {
	// Launch a kernel on the GPU with one thread for each element.
	distanceKernel << < dim3((unsigned)ceil(n / CUDA_THREADS_PER_BLOCK), (unsigned)pointsCount), (unsigned)CUDA_THREADS_PER_BLOCK >> >(dev_points, dev_distances, n, startPoint);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "distanceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	return cudaStatus;
}

/**
 * Retrieves results of last distances calculation on cuda.
 * @param double* distances Results destination
 * @param int n
 * @param int pointsCount
 * @returns int Cuda status
 */
int cudaResult(double* distances, int n, int pointsCount) {
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return 1;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(distances, dev_distances, pointsCount * n * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 1;
	}

	return cudaStatus;
}

/**
 * Releases allocated memory and finalizes GPU.
 */
int cudaFinalize() {
	hipFree(dev_points);
	hipFree(dev_distances);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return cudaStatus;
}
